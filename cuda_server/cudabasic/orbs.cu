//%%writefile Orbs.cu
//This file is tested successfully on Google Colab with V100 GPU.
//cps:8.269329e+11 in Google Colab with V100 GPU
//cps:1.588200e+13 in Google Colab with A100 GPU
//cps:3.656473e+11 in Google Colab with A100 GPU
//cps:3.549229e+10 in Google Colab with T4 GPU, more size of orbs will run faster.
#include<stdio.h>
#include<stdlib.h>
#include<hip/hip_runtime.h>
#include<time.h>
//#include<math.h> //cuda already has sqrt

struct Orb {
  double x;
  double y;
  double z;
  double vx;
  double vy;
  double vz;
  double mass;
  int id;
};

const double G  = 0.000005;
const double SPEED_LIMIT = 4.0;
const double MIN_DIST = 0.5;
const double PI = 3.14159265358979323846;

__device__ void OrbUpdate(Orb *o, Orb *oList, int nOrb) {
  if (o->id > 0) {
    //double aAll = CalcGravityAll()
    double gAllx = 0, gAlly = 0, gAllz = 0;
    for (int i=0; i<nOrb; ++i) {
      Orb *target = &oList[i];
      if (target->id < 0 || target->id == o->id) {
        return;
      }
      double distSq = (o->x-target->x)*(o->x-target->x) + (o->y-target->y)*(o->y-target->y) + (o->z-target->z)*(o->z-target->z);
      double dist = sqrt(distSq);

      // if tooNearly or overSpeeded
      if (dist < MIN_DIST) {
        o->id = - o->id; // mark status
        // here need transfer mass to target
        target->mass += o->mass; // will cause concurrency problem.
        o->mass = 0.000000001;
        printf("Orb(%d) got crashed by dist\n", o->id);
        return;
      }

      double gTar = target->mass / distSq * G;
      gAllx += -gTar * (o->x-target->x) / dist;
      gAlly += -gTar * (o->y-target->y) / dist;
      gAllz += -gTar * (o->z-target->z) / dist;
    }

    o->x += o->vx;
    o->y += o->vy;
    o->z += o->vz;
    o->vx += gAllx;
    o->vy += gAlly;
    o->vz += gAllz;

    if (o->vx > SPEED_LIMIT || o->vy > SPEED_LIMIT || o->vz > SPEED_LIMIT) {
      o->id = - o->id;
      printf("Orb(%d) get crashed by overspeed\n", o->id);
      return;
    }
  }
}


__device__ void UpdateOrbList(Orb *oList, int nOrb) {
  for (int i=0; i<nOrb; ++i) {
      Orb *o = &oList[i];
      OrbUpdate(o, oList, nOrb);
  }
  // should clear when orb.id < 0
}

__global__ void UpdateOrbs(Orb *oList, int nOrb, int nTimes) {
  for (int i=0; i<nTimes; ++i) {
    UpdateOrbList(oList, nOrb);
  }
}

const double MASS_RANGE = 1;
const double DISTRI_WIDE = 10000;
const double VELO_RANGE = 0.005;

int main()
{
    int nOrb = 10000;
    int nTimes = 10000;
    
    // 申请host内存
    Orb *oList = (Orb*)malloc(nOrb * sizeof(Orb));

    // 初始化数据
    for (int i = 0; i < nOrb; ++i) {
      oList[i].id = i+1;
      oList[i].mass = (double)rand() / RAND_MAX * MASS_RANGE;
      double radius = DISTRI_WIDE * (double)rand() / RAND_MAX;
      double idx = (double)rand() / RAND_MAX * PI * 2;
      oList[i].x = cos(idx) * radius;
      oList[i].y = sin(idx) * radius;
      oList[i].z = (double)rand() / RAND_MAX - 0.5;
      oList[i].vx = cos(idx+PI/2.0) * VELO_RANGE;
      oList[i].vy = sin(idx+PI/2.0) * VELO_RANGE;
      //printf("[%f,%f,%f,%f,%f,%f,%f,%d]\n", oList[i].x, oList[i].y, oList[i].z, oList[i].vz, oList[i].vy, oList[i].vz, oList[i].mass, oList[i].id);
    }

    printf("init ok, nOrb:%d nTimes:%d, will times:%ld\n", nOrb, nTimes, long(nOrb)*long(nOrb)*long(nTimes));
    clock_t timeStart = clock();

    // 申请device内存
    Orb *doList;
    hipMalloc((void**)&doList, nOrb*sizeof(Orb));

    // 将host数据拷贝到device
    hipMemcpy((void*)doList, (void*)oList, nOrb*sizeof(Orb), hipMemcpyHostToDevice);
    // 定义kernel的执行配置
    //dim3 blockSize(256);
    //dim3 gridSize((nOrb + blockSize.x - 1) / blockSize.x);
    // 执行kernel
    //UpdateOrbs <<< gridSize, blockSize >>>(oList, nOrb, nTimes);
    UpdateOrbs <<< 32, 32 >>>(oList, nOrb, nTimes);
    //UpdateOrbs(oList, nOrb, nTimes); // use host

    // 将device得到的结果拷贝到host
    hipMemcpy((void*)oList, (void*)doList, nOrb*sizeof(Orb), hipMemcpyDeviceToHost);

    clock_t timeEnd = clock();

    // 检查执行结果
    printf("all done. nOrb:%d times:%ld use time:%f cps:%e\n", 
      nOrb, 
      long(nOrb)*long(nOrb)*long(nTimes), 
      double(timeEnd-timeStart)/CLOCKS_PER_SEC, 
      double(long(nOrb)*long(nOrb)*long(nTimes))/(double(timeEnd-timeStart)/CLOCKS_PER_SEC));

    // 释放device内存
    hipFree(doList);
    // 释放host内存
    free(oList);

    return 0;
}