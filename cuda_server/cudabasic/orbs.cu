//%%writefile Orbs.cu
//This file is tested successfully on Google Colab with V100 GPU.
//cps:8.269329e+11 in Google Colab with V100 GPU
//cps:1.588200e+13 in Google Colab with A100 GPU
//cps:3.656473e+11 in Google Colab with A100 GPU
//cps:3.549229e+10 in Google Colab with T4 GPU, more size of orbs will run faster.
// build: nvcc -o Orbs Orbs.cu -lm
#include<stdio.h>
#include<stdlib.h>
#include<hip/hip_runtime.h>
#include<time.h>
//#include<math.h> //cuda already has sqrt

struct Orb {
  double x;
  double y;
  double z;
  double vx;
  double vy;
  double vz;
  double mass;
  int id;
};

const double PI = 3.14159265358979323846;
const double G  = 0.0005;
const double SPEED_LIMIT = 4.0;
const double MIN_DIST = 0.5;
const double MASS_RANGE = 100;
const double DISTRI_WIDE = 10000;
const double VELO_RANGE = 0.05;

__device__ void OrbUpdate(Orb *o, Orb *oList, int nOrb) {
  //printf("DEBUG:"__FILE__":%d id:%d tid:%d\n",__LINE__,o->id,threadIdx.x + blockIdx.x * blockDim.x);
  if (o->id > 0) {
    double gAllx = 0, gAlly = 0, gAllz = 0;
    for (int i=0; i<nOrb; ++i) {
      Orb *target = &oList[i];
      if (target->id < 0 || target->id == o->id) {
        continue;
      }
      double distSq = (o->x-target->x)*(o->x-target->x) + (o->y-target->y)*(o->y-target->y) + (o->z-target->z)*(o->z-target->z);
      double dist = sqrt(distSq);

      // if tooNearly or overSpeeded
      if (dist < MIN_DIST) {
        o->id = - o->id; // mark status
        // here need transfer mass to target
        target->mass += o->mass; // will cause concurrency problem.
        o->mass = 0.000000001;
        printf("Orb(%d) got crashed by dist too close\n", o->id);
        break;
      }
      
      double gTar = target->mass / distSq * G;
      gAllx += -gTar * (o->x-target->x) / dist;
      gAlly += -gTar * (o->y-target->y) / dist;
      gAllz += -gTar * (o->z-target->z) / dist;
    }
    
    o->x += o->vx;
    o->y += o->vy;
    o->z += o->vz;
    o->vx += gAllx;
    o->vy += gAlly;
    o->vz += gAllz;

    //printf("\t[%f,%f,%f,%f,%f,%f,%f,%d](%e,%e,%e)\n", o->x, o->y, o->z, o->vx, o->vy, o->vz, o->mass, o->id, gAllx, gAlly, gAllz);
    if (o->vx > SPEED_LIMIT || o->vy > SPEED_LIMIT || o->vz > SPEED_LIMIT) {
      o->id = - o->id;
      printf("Orb(%d) get crashed by overspeed\n", o->id);
      return;
    }
  }
}

__global__ void UpdateOrbList(Orb *oList, int nOrb) {
  for (int i=0; i<nOrb; ++i) {
      OrbUpdate(&oList[i], oList, nOrb);
  }
}

__global__ void ThreadUpdateOrb(Orb *oList, int nOrb) {
  int i = threadIdx.x + blockIdx.x * blockDim.x;
  if (i < nOrb) {
      OrbUpdate(&oList[i], oList, nOrb);
  //} else { printf("the i exceeded:%d\n", i); // realy will be exceeded if thread/block too more than nOrb
  }
}

void PrintOrbList(Orb *oList, int nOrb) {
  for (int i=0; i<nOrb; ++i) {
      printf("[%f,%f,%f,%e,%e,%e,%f,%d]\n", oList[i].x, oList[i].y, oList[i].z, oList[i].vx, oList[i].vy, oList[i].vz, oList[i].mass, oList[i].id);
  }
}

void DiffOrbList(Orb *oList, int nOrb, Orb *oListDiff) {
  for (int i=0; i<nOrb; ++i) {
    oListDiff[i].x -= oList[i].x;
    oListDiff[i].y -= oList[i].y;
    oListDiff[i].z -= oList[i].z;
    oListDiff[i].vx -= oList[i].vx;
    oListDiff[i].vy -= oList[i].vy;
    oListDiff[i].vz -= oList[i].vz;
    oListDiff[i].mass -= oList[i].mass;
  }
}

int main()
{
    int nOrb = 3;
    int nTimes = 40000;
    srand(time(NULL));
    
    // 申请host内存
    Orb *oList = (Orb*)malloc(nOrb * sizeof(Orb));
    Orb *oList2 = (Orb*)malloc(nOrb * sizeof(Orb));

    // 初始化数据
    for (int i = 0; i < nOrb; ++i) {
      oList[i].id = i+1;
      oList[i].mass = (double)rand() / RAND_MAX * MASS_RANGE;
      double radius = DISTRI_WIDE * (double)rand() / RAND_MAX;
      double idx = (double)rand() / RAND_MAX * PI * 2;
      oList[i].x = cos(idx) * radius;
      oList[i].y = sin(idx) * radius;
      oList[i].z = (double)rand() / RAND_MAX - 0.5;
      oList[i].vx = cos(idx+PI/2.0) * VELO_RANGE;
      oList[i].vy = sin(idx+PI/2.0) * VELO_RANGE;
    }

    printf("init ok, nOrb:%d nTimes:%d, will times:%ld\n", nOrb, nTimes, long(nOrb)*long(nOrb)*long(nTimes));
    PrintOrbList(oList, nOrb);
    clock_t timeStart = clock();

    // 申请device内存
    Orb *doList;
    hipMalloc((void**)&doList, nOrb*sizeof(Orb));

    // 将host数据拷贝到device
    hipMemcpy((void*)doList, (void*)oList, nOrb*sizeof(Orb), hipMemcpyHostToDevice);
    
    // 定义kernel的执行配置
    dim3 blockSize(256);
    dim3 gridSize((nOrb + blockSize.x - 1) / blockSize.x);
    // 执行kernel
    //UpdateOrbs <<< gridSize, blockSize >>>(doList, nOrb, nTimes);
    for (int i=0; i<nTimes; ++i) {
      ThreadUpdateOrb <<< gridSize, blockSize >>>(doList, nOrb);
      hipDeviceSynchronize(); //调用次数越少越好
      if (nTimes >= 10 && (i+1)%(nTimes/10) == 0) {
        printf("times process:%d/%d\n", i, nTimes);
        hipMemcpy((void*)oList2, (void*)doList, nOrb*sizeof(Orb), hipMemcpyDeviceToHost);
        PrintOrbList(oList2, nOrb);
      }
    }

    // 将device得到的结果拷贝到host
    hipMemcpy((void*)oList2, (void*)doList, nOrb*sizeof(Orb), hipMemcpyDeviceToHost);

    clock_t timeEnd = clock();

    // 检查执行结果
    printf("all done. nOrb:%d times:%ld use time:%f cps:%e\n", 
      nOrb, 
      long(nOrb)*long(nOrb)*long(nTimes), 
      double(timeEnd-timeStart)/CLOCKS_PER_SEC, 
      double(long(nOrb)*long(nOrb)*long(nTimes))/(double(timeEnd-timeStart)/CLOCKS_PER_SEC));
    
    DiffOrbList(oList, nOrb, oList2);
    PrintOrbList(oList2, nOrb);

    // 释放device内存 & 释放host内存
    hipFree(doList);
    free(oList);
    free(oList2);

    return 0;
}