//%%writefile Orbs.cu
//This file is tested successfully on Google Colab with V100 GPU.
//cps:8.269329e+11 in Google Colab with V100 GPU
//cps:3.656473e+11 in Google Colab with A100 GPU
//cps:3.549229e+10 in Google Colab with T4 GPU, more size of orbs will run faster.
// build: nvcc -o Orbs Orbs.cu -lm -lpthread
#include<stdio.h>
#include<stdlib.h>
#include<hip/hip_runtime.h>
#include<time.h>
#include<unistd.h>
#include<pthread.h>

struct Orb {
  double x;
  double y;
  double z;
  double vx;
  double vy;
  double vz;
  double mass;
  int id;
};

struct OrbList {
  Orb *list;
  int n;
};
struct SavingThreadParam {
  Orb *list;
  int n;
  int state; //1==running, 0==stop
  pthread_t tid;
};

const double PI = 3.14159265358979323846;
const double G  = 0.00005;
const double SPEED_LIMIT = 4.0;
const double MIN_DIST = 0.5;
const double MASS_RANGE = 100;
const double DISTRI_WIDE = 10000;
const double VELO_RANGE = 0.005;

__device__ void OrbUpdate(Orb *o, Orb *oList, int nOrb) {
  if (o->id > 0) {
    double gAllx = 0, gAlly = 0, gAllz = 0; // double3 gAll = {0, 0, 0};
    for (int i=0; i<nOrb; ++i) {
      Orb *target = &oList[i];
      if (target->id < 0 || target->id == o->id) {
        continue;
      }

      double distSq = (o->x-target->x)*(o->x-target->x) + (o->y-target->y)*(o->y-target->y) + (o->z-target->z)*(o->z-target->z);
      // if tooNearly or overSpeeded
      if (distSq < MIN_DIST*MIN_DIST) {
        o->id = - o->id; // mark status
        //target->mass += o->mass; // transfer mass to target, will cause concurrency problem.
        //o->mass = 0.000000001;
        //printf("%d crashed by MIN_DIST\n", o->id);
        break;
      }
      
      double rdist = rsqrt(distSq);
      double gTar = target->mass / distSq * G;
      gAllx += -gTar * (o->x-target->x) * rdist;
      gAlly += -gTar * (o->y-target->y) * rdist;
      gAllz += -gTar * (o->z-target->z) * rdist;
    }
    
    o->x += o->vx;
    o->y += o->vy;
    o->z += o->vz;
    o->vx += gAllx;
    o->vy += gAlly;
    o->vz += gAllz;

    if (o->vx > SPEED_LIMIT || o->vy > SPEED_LIMIT || o->vz > SPEED_LIMIT) {
      o->id = - o->id;
      //printf("%d crashed by overspeed\n", o->id);
      return;
    }
  }
}

__global__ void ThreadUpdateOrb(Orb *oList, int nOrb) {
  int i = threadIdx.x + blockIdx.x * blockDim.x;
  if (i < nOrb) {
      OrbUpdate(&oList[i], oList, nOrb);
  //} else { printf("the i exceeded:%d\n", i); // realy will be exceeded if thread/block too more than nOrb
  }
}

void PrintOrbList(Orb *oList, int nOrb) {
  for (int i=0; i<nOrb; ++i) {
      printf("[%f,%f,%f,%e,%e,%e,%f,%d]\n", oList[i].x, oList[i].y, oList[i].z, oList[i].vx, oList[i].vy, oList[i].vz, oList[i].mass, oList[i].id);
  }
}

void DiffOrbList(Orb *oList, int nOrb, Orb *oListDiff) {
  Orb oSum;
  for (int i=0; i<nOrb; ++i) {
    oSum.x += oListDiff[i].x - oList[i].x;
    oSum.y += oListDiff[i].y - oList[i].y;
    oSum.z += oListDiff[i].z - oList[i].z;
    oSum.vx += oListDiff[i].vx - oList[i].vx;
    oSum.vy += oListDiff[i].vy - oList[i].vy;
    oSum.vz += oListDiff[i].vz - oList[i].vz;
    oSum.mass += oListDiff[i].mass - oList[i].mass;
  }
  oSum.x /= double(nOrb);
  oSum.y /= double(nOrb);
  oSum.z /= double(nOrb);
  oSum.vx /= double(nOrb);
  oSum.vy /= double(nOrb);
  oSum.vz /= double(nOrb);
  oSum.mass /= double(nOrb);
  printf("avg diff:%g,%g,%g,%g,%g,%g,%g\n", oSum.x, oSum.y, oSum.z, oSum.vx, oSum.vy, oSum.vz, oSum.mass);
}

void SaveOrbList(Orb *oList, int nOrb, const char* filename) {
  FILE* f = fopen(filename, "w");
  if (f == NULL) {
    printf("Error opening file %s!\n", filename);
    return;
  }
  fputs("[", f);
  for (int i=0; i<nOrb; ++i) {
      fprintf(f, "[%.15g,%.15g,%.15g,%.15g,%.15g,%.15g,%g,%d]", oList[i].x, oList[i].y, oList[i].z, oList[i].vx, oList[i].vy, oList[i].vz, oList[i].mass, oList[i].id);
      if (i < nOrb-1) {
        fputs(",", f);
      }
  }
  fputs("]", f);
  fclose(f);
}

const char *loadFile = "";
const char *saveFile = "list1.json";

// need exclusive parameter
void* ThreadSavingOrbList(void* ptr) {
  //OrbList *oList = (OrbList*)ptr;
  SavingThreadParam *param = (SavingThreadParam*)ptr;
  while (param->state == 1) {
    usleep(500000);
    SaveOrbList(param->list, param->n, saveFile);
  }
  return NULL;
}

Orb *LoadOrbList(const char* loadFile, int *nOrbLoaded) {
    Orb *oList = NULL;
    int nOrb = 0;
    FILE* f = fopen(loadFile, "r");
    if (f == NULL) {
      printf("Error loading file %s!\n", loadFile);
      return NULL;
    }
    // read file and count the orbs
    char buf[256] = "";
    int bracketIndent = 0;
    while (fgets(buf, 256, f) != NULL) {
      for (int i=0; i<256 && buf[i] != '\0'; ++i) {
          bracketIndent += buf[i] == '[' ? 1 : 0;
          bracketIndent -= buf[i] == ']' ? 1 : 0;
          if (buf[i] == '[') {
            nOrb += 1;
          }
      }
    }
    nOrb--;
    printf("according to loadFile, nOrb:%d lastIndent:%d\n", nOrb, bracketIndent);
    if (nOrb <= 0 || bracketIndent != 0) {
      printf("file content error! no orbs loaded\n");
      fclose(f);
      return NULL;
    }
    oList = (Orb*)malloc(nOrb * sizeof(Orb));

    rewind(f);
    bracketIndent = 0;
    int orbIdx = 0;
    char restLine[512] = "";
    while (fgets(buf, 256, f) != NULL) {
      strcat(restLine, buf);
      int lastLeftBracket = -1;
      //printf("the restLine len:%d we will handle:<<%s>>\n", strlen(restLine), restLine);
      for (int i=0; i<512 && restLine[i] != '\0'; ++i) {
          if (restLine[i] == '[') {
            bracketIndent += 1;
            lastLeftBracket = i;
            //printf("find [ at:%d bracketIndent:%d\n", lastLeftBracket, bracketIndent);
          }
          if (restLine[i] == ']') {
            bracketIndent -= 1;
            //printf("find ] at:%d bracketIndent:%d\n", lastRightBracket, bracketIndent);
            if (bracketIndent == 1) {
              
              // 扫到右括号才开始解析
              sscanf(restLine+lastLeftBracket+1, "%lf,%lf,%lf,%lf,%lf,%lf,%lf,%d", 
                &oList[orbIdx].x, &oList[orbIdx].y, &oList[orbIdx].z, &oList[orbIdx].vx, &oList[orbIdx].vy, &oList[orbIdx].vz, &oList[orbIdx].mass, &oList[orbIdx].id);
              ;
	            //printf("loaded orb:%e,%e,%e,%e,%e,%e,%e,%d\n", oList[orbIdx].x, oList[orbIdx].y, oList[orbIdx].z, oList[orbIdx].vx, oList[orbIdx].vy, oList[orbIdx].vz, oList[orbIdx].mass, oList[orbIdx].id);
              orbIdx += 1;
            }
          }
      }
      if (bracketIndent == 2) {
        strcpy(restLine, restLine+lastLeftBracket+1);
      } else {
        restLine[0] = '\0';
      }
      //printf("bracketIndent:%d [ at:%d ] at:%d restLine:%s\n", bracketIndent, lastLeftBracket, lastRightBracket, restLine);
    }
    fclose(f);
    *nOrbLoaded = nOrb;
    return oList;
}

// ./Orbs -n 3 -t 40000 -l list1.json -s list1.json
int main(int argc, char *argv[]) {
    int nOrb = 3;
    int nTimes = 40000;
    srand(time(NULL));

    // Parse arguments
    if (argc >= 2) {
      for (int i = 0; i < argc; ++i) {
        if (strcmp(argv[i], "-n") == 0 && i+1 < argc) {
            nOrb = atoi(argv[i+1]);
        }
        if (strcmp(argv[i], "-t") == 0 && i+1 < argc) {
            nTimes = atoi(argv[i+1]);
        }
        if (strcmp(argv[i], "-l") == 0 && i+1 < argc) {
            loadFile = argv[i+1];
        }
        if (strcmp(argv[i], "-s") == 0 && i+1 < argc) {
            saveFile = argv[i+1];
        }
      }
    }

    // 申请host内存
    Orb *oList;// = (Orb*)malloc(nOrb * sizeof(Orb));
    Orb *oList2;// = (Orb*)malloc(nOrb * sizeof(Orb));

    // 初始化数据
    if (strcmp(loadFile, "") == 0) {
      oList = (Orb*)malloc(nOrb * sizeof(Orb));
      oList2 = (Orb*)malloc(nOrb * sizeof(Orb));
      for (int i = 0; i < nOrb; ++i) {
        oList[i].id = i+1;
        oList[i].mass = (double)rand() / RAND_MAX * MASS_RANGE;
        double radius = DISTRI_WIDE * (double)rand() / RAND_MAX;
        double idx = (double)rand() / RAND_MAX * PI * 2;
        oList[i].x = cos(idx) * radius;
        oList[i].y = sin(idx) * radius;
        oList[i].z = ((double)rand() / RAND_MAX - 0.5)*2*DISTRI_WIDE/1000;
        oList[i].vx = cos(idx+PI/2.0) * VELO_RANGE;
        oList[i].vy = sin(idx+PI/2.0) * VELO_RANGE;
      }
      memcpy(oList2, oList, nOrb*sizeof(Orb));
    } else {
      // load file from json
      oList = LoadOrbList(loadFile, &nOrb);
      if (oList == NULL) {
	 printf("load from loadFile %s failed\n", loadFile);
	 return 0;
      }
      oList2 = (Orb*)malloc(nOrb *sizeof(Orb));
      memcpy(oList2, oList, nOrb*sizeof(Orb));
    }
    //PrintOrbList(oList, nOrb);

    // 申请device内存
    Orb *doList;
    hipMalloc((void**)&doList, nOrb*sizeof(Orb));

    // 将host数据拷贝到device
    hipMemcpy((void*)doList, (void*)oList, nOrb*sizeof(Orb), hipMemcpyHostToDevice);
    
    // 定义kernel的执行配置 // only 1024 work well
    dim3 blockSize(1024);
    dim3 gridSize((nOrb + blockSize.x - 1) / blockSize.x);

    printf("init ok, nOrb:%d nTimes:%d, will times:%ld loadFile:%s gridSize:%d blockSize:%d\n", nOrb, nTimes, long(nOrb)*long(nOrb)*long(nTimes), loadFile, gridSize.x, blockSize.x);

    // Start a thread to save orb list
    SavingThreadParam param = {oList2, nOrb, 1};
    pthread_create(&param.tid, NULL, ThreadSavingOrbList, &param);

    clock_t timeStart = clock();

    // 执行kernel
    for (int i=0; i<nTimes; ++i) {
      ThreadUpdateOrb <<< gridSize, blockSize >>>(doList, nOrb);
      hipDeviceSynchronize(); //调用次数越少越好
      if (nTimes >= 10 && (i+1)%(nTimes/10) == 0) {
        printf("process:%d/%d, time:%.3f cps:%e estimate remain:%.3fs\n", i+1, nTimes, (double(clock()-timeStart)/CLOCKS_PER_SEC), double(long(nOrb)*long(nOrb)*long(i+1))/(double(clock()-timeStart)/CLOCKS_PER_SEC), double(nTimes-i-1)/double(i+1)*(double(clock()-timeStart)/CLOCKS_PER_SEC));
        hipMemcpy((void*)oList2, (void*)doList, nOrb*sizeof(Orb), hipMemcpyDeviceToHost);
      }
    }

    param.state = 0; // stop the thread
    // 将device得到的结果拷贝到host
    hipMemcpy((void*)oList2, (void*)doList, nOrb*sizeof(Orb), hipMemcpyDeviceToHost);

    clock_t timeEnd = clock();
    DiffOrbList(oList, nOrb, oList2);
    SaveOrbList(oList2, nOrb, saveFile);

    // 检查执行结果
    printf("all done. nOrb:%d times:%ld use time:%f cps:%e\n", 
      nOrb, 
      long(nOrb)*long(nOrb)*long(nTimes), 
      double(timeEnd-timeStart)/CLOCKS_PER_SEC, 
      double(long(nOrb)*long(nOrb)*long(nTimes))/(double(timeEnd-timeStart)/CLOCKS_PER_SEC));
    
    // 释放device内存 & 释放host内存
    hipFree(doList);
    free(oList);
    free(oList2);

    return 0;
}
